#include "hip/hip_runtime.h"
#include "../include/somigliana/cuda_impl.h"

#include <Eigen/Dense>

typedef Eigen::Matrix<scalar_t, 3, 1> Vec3;
typedef Eigen::Matrix<scalar_t, 3, 3> Mat3;

#define CUDA_PI 3.141592653589793f

inline __device__ int sgn(const scalar_t val) {
  return ((0.0 < val)-(val < 0.0));
}

inline __device__
scalar_t double_trig_area(const Vec3 &a, const Vec3 &b, const Vec3 &c) {
  return ((b-a).cross(c-a)).norm();
}

__global__ void mvc_kernel(scalar_t *d_PHI,
                           const scalar_t *d_V,
                           const index_t  *d_cageF,
                           const scalar_t *d_cageV,
                           const index_t nv,
                           const index_t ncf,
                           const index_t ncv) {
  unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

  if ( index < nv ) {
    Vec3 x(d_V[3*index+0], d_V[3*index+1], d_V[3*index+2]);

    // for each face
    for (index_t f = 0; f < ncf; ++f) {
      index_t vx = d_cageF[3*f+0], vy = d_cageF[3*f+1], vz = d_cageF[3*f+2];

      Mat3 u;
      {
        u.col(0) = Vec3(d_cageV[3*vx+0], d_cageV[3*vx+1], d_cageV[3*vx+2])-x;
        u.col(1) = Vec3(d_cageV[3*vy+0], d_cageV[3*vy+1], d_cageV[3*vy+2])-x;
        u.col(2) = Vec3(d_cageV[3*vz+0], d_cageV[3*vz+1], d_cageV[3*vz+2])-x;
      }
      Vec3 d(u.col(0).norm(), u.col(1).norm(), u.col(2).norm());
      u.colwise().normalize();
      scalar_t sgn_u = sgn(u.determinant());
      
      Vec3 l, theta, c, s;
      {
        l[0] = (u.col(1)-u.col(2)).norm();
        l[1] = (u.col(2)-u.col(0)).norm();
        l[2] = (u.col(0)-u.col(1)).norm();

        theta[0] = 2*asin(l[0]/2);
        theta[1] = 2*asin(l[1]/2);
        theta[2] = 2*asin(l[2]/2);
      }

      const scalar_t h = theta.sum()/2;      

      const scalar_t TOL = 1e-8;
      if ( fabs(CUDA_PI-h) < TOL ) {
        d_PHI[vx+ncv*index] += sin(theta[0])*d[1]*d[2];
        d_PHI[vy+ncv*index] += sin(theta[1])*d[2]*d[0];
        d_PHI[vz+ncv*index] += sin(theta[2])*d[0]*d[1];
      } else {        
        c[0] = 2*sin(h)*sin(h-theta[0])
            /(sin(theta[1])*sin(theta[2]))-1;
        c[1] = 2*sin(h)*sin(h-theta[1])
            /(sin(theta[2])*sin(theta[0]))-1;
        c[2] = 2*sin(h)*sin(h-theta[2])
            /(sin(theta[0])*sin(theta[1]))-1; 

        s[0] = sgn_u*sqrt(1-c[0]*c[0]);
        s[1] = sgn_u*sqrt(1-c[1]*c[1]);
        s[2] = sgn_u*sqrt(1-c[2]*c[2]);

        if ( fabs(s[0]) < TOL || fabs(s[1]) < TOL || fabs(s[2]) < TOL ) {
          continue;
        }
        
        // assign
        d_PHI[vx+ncv*index] += (theta[0]-c[2]*theta[1]-c[1]*theta[2])/(d[0]*sin(theta[2])*s[1]); 
        d_PHI[vy+ncv*index] += (theta[1]-c[0]*theta[2]-c[2]*theta[0])/(d[1]*sin(theta[0])*s[2]); 
        d_PHI[vz+ncv*index] += (theta[2]-c[1]*theta[0]-c[0]*theta[1])/(d[2]*sin(theta[1])*s[0]); 
      }      
    } // end for f    
  } // end if index...
}

__global__ void green_kernel(scalar_t *d_phix,
                             scalar_t *d_phiy,
                             scalar_t *d_phiz,
                             scalar_t *d_psi,
                             const scalar_t *d_V,
                             const index_t  *d_cageF,
                             const scalar_t *d_cageV,
                             const scalar_t *d_cageN,
                             const index_t nv,
                             const index_t ncf,
                             const index_t ncv,
                             const scalar_t *d_qp,
                             const scalar_t *d_qw,
                             const index_t nq) {
  unsigned int thread_index = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int index = thread_index/ncf, f = thread_index%ncf;

  if ( f < ncf && index < nv ) {
    const index_t ix = d_cageF[3*f+0], iy = d_cageF[3*f+1], iz = d_cageF[3*f+2];
    Vec3 x (d_V[3*index+0],  d_V[3*index+1],  d_V[3*index+2]);
    Vec3 v0(d_cageV[3*ix+0], d_cageV[3*ix+1], d_cageV[3*ix+2]);
    Vec3 v1(d_cageV[3*iy+0], d_cageV[3*iy+1], d_cageV[3*iy+2]);
    Vec3 v2(d_cageV[3*iz+0], d_cageV[3*iz+1], d_cageV[3*iz+2]);
    Vec3 n (d_cageN[3*f+0],  d_cageN[3*f+1],  d_cageN[3*f+2]);   
    const scalar_t absA = fabs(double_trig_area(v0, v1, v2));

    Vec3 v, w, d;
    {
      v = v1-v0;
      w = v2-v0;
      d = v0-x;
    }

    const scalar_t C0 = 1/(4*CUDA_PI);
    
    scalar_t psi, phi0, phi1, phi2;
    psi = phi0 = phi1 = phi2 = 0;
    for (index_t i = 0; i < nq; ++i) {
      const scalar_t a = d_qp[2*i+0], b = d_qp[2*i+1], weight = d_qw[i];
      const Vec3 r = d+a*v+b*w;
      const scalar_t i_rn = 1.0/r.norm(), i_rn3 = i_rn*i_rn*i_rn;

      scalar_t G = C0*i_rn;
      scalar_t dG = C0*i_rn3*r.dot(n);
      
      psi  += weight*G;
      phi0 += weight*(1-a-b)*dG;
      phi1 += weight*a*dG;
      phi2 += weight*b*dG;
    }

    // assign
    d_psi [f+ncf*index] = absA*psi;
    d_phix[f+ncf*index] = absA*phi0;
    d_phiy[f+ncf*index] = absA*phi1;
    d_phiz[f+ncf*index] = absA*phi2;
  } // end if index
}

__global__ void green_kernel_post(scalar_t *d_phi,
                                  const scalar_t *d_phix,
                                  const scalar_t *d_phiy,
                                  const scalar_t *d_phiz,
                                  const index_t *d_cageF,
                                  const index_t nv,
                                  const index_t ncf,
                                  const index_t ncv) {
  unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

  if ( index < nv ) {
    for (index_t f = 0; f < ncf; ++f) { // for each face
      d_phi[ncv*index+d_cageF[3*f+0]] += d_phix[ncf*index+f];
      d_phi[ncv*index+d_cageF[3*f+1]] += d_phiy[ncf*index+f];
      d_phi[ncv*index+d_cageF[3*f+2]] += d_phiz[ncf*index+f];      
    } // endfor
  } // endif 
}

__global__ void somig_kernel(const scalar_t nu,
                             scalar_t *d_PHIx,
                             scalar_t *d_PHIy,
                             scalar_t *d_PHIz,
                             scalar_t *d_PSI ,
                             const scalar_t *d_V,
                             const index_t  *d_cageF,
                             const scalar_t *d_cageV,
                             const scalar_t *d_cageN,
                             const index_t nv,
                             const index_t ncf,
                             const index_t ncv,
                             const scalar_t *d_qp,
                             const scalar_t *d_qw,
                             const index_t nq)  {
  unsigned int thread_index = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int index = thread_index/ncf, f = thread_index%ncf;

  if ( f < ncf && index < nv ) {
    const scalar_t mu = 1e0;
    const scalar_t C1 = 1.0/(16*CUDA_PI*mu*(1-nu));
    const scalar_t C2 = -1.0/(8*CUDA_PI*(1-nu));
    const Mat3 Id = Mat3::Identity();
    
    const index_t ix = d_cageF[3*f+0], iy = d_cageF[3*f+1], iz = d_cageF[3*f+2];
    Vec3 x (d_V[3*index+0],  d_V[3*index+1],  d_V[3*index+2]);
    Vec3 v0(d_cageV[3*ix+0], d_cageV[3*ix+1], d_cageV[3*ix+2]);
    Vec3 v1(d_cageV[3*iy+0], d_cageV[3*iy+1], d_cageV[3*iy+2]);
    Vec3 v2(d_cageV[3*iz+0], d_cageV[3*iz+1], d_cageV[3*iz+2]);
    Vec3 n (d_cageN[3*f+0],  d_cageN[3*f+1],  d_cageN[3*f+2]);

    const scalar_t absA = fabs(double_trig_area(v0, v1, v2));

    Vec3 v, w, d;
    {
      v = v1-v0;
      w = v2-v0;
      d = v0-x;
    }

    const scalar_t c_12nu = 1-2*nu, c_34nu = 3-4*nu;
    
    Mat3 psi, phi0, phi1, phi2;
    psi = phi0 = phi1 = phi2 = Mat3::Zero();
    Mat3 rrT, tmp;
    for (index_t i = 0; i < nq; ++i) {
      const scalar_t a = d_qp[2*i+0], b = d_qp[2*i+1], weight = d_qw[i];
      const Vec3 r = d+a*v+b*w;
      const scalar_t i_rn = 1.0/r.norm(), i_rn3 = i_rn*i_rn*i_rn,
          i_rn5 = i_rn3*i_rn*i_rn;

      rrT = r*r.transpose();
      tmp = (c_12nu*r.dot(n)*i_rn3*Id+
             3*r.dot(n)*i_rn5*rrT-
             c_12nu*i_rn3*(r*n.transpose()-n*r.transpose()));            
      
      psi  += weight*(c_34nu*i_rn*Id+rrT*i_rn3);
      phi0 += weight*(1-a-b)*tmp;
      phi1 += weight*a*tmp;
      phi2 += weight*b*tmp;
    }

    psi  *=  C1*absA;
    phi0 *= -C2*absA;
    phi1 *= -C2*absA;
    phi2 *= -C2*absA;

    scalar_t* ptr[4] = {d_PSI, d_PHIx, d_PHIy, d_PHIz};
    Mat3*     cof[4] = {&psi,  &phi0,  &phi1,  &phi2};
    for (index_t i = 0; i < 4; ++i) {
      *(ptr[i]+3*nv*(3*f+0)+3*index+0) += (*cof[i])(0, 0);
      *(ptr[i]+3*nv*(3*f+0)+3*index+1) += (*cof[i])(1, 0);
      *(ptr[i]+3*nv*(3*f+0)+3*index+2) += (*cof[i])(2, 0);

      *(ptr[i]+3*nv*(3*f+1)+3*index+0) += (*cof[i])(0, 1); 
      *(ptr[i]+3*nv*(3*f+1)+3*index+1) += (*cof[i])(1, 1); 
      *(ptr[i]+3*nv*(3*f+1)+3*index+2) += (*cof[i])(2, 1); 

      *(ptr[i]+3*nv*(3*f+2)+3*index+0) += (*cof[i])(0, 2);
      *(ptr[i]+3*nv*(3*f+2)+3*index+1) += (*cof[i])(1, 2);
      *(ptr[i]+3*nv*(3*f+2)+3*index+2) += (*cof[i])(2, 2);
    }
  } // end if
}

__global__ void somig_kernel_post(scalar_t *d_PHI,
                                  const scalar_t *d_PHIx,
                                  const scalar_t *d_PHIy,
                                  const scalar_t *d_PHIz,
                                  const index_t *d_cageF,
                                  const index_t nv,
                                  const index_t ncf) {
  unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

  if ( index < nv ) {
    const scalar_t* ptr[3] = {d_PHIx, d_PHIy, d_PHIz};

    for (index_t f = 0; f < ncf; ++f) { // for each face
      for (index_t i = 0; i < 3; ++i) {
        const index_t v = d_cageF[3*f+i];
        d_PHI[3*nv*(3*v+0)+3*index+0] += ptr[i][3*nv*(3*f+0)+3*index+0];
        d_PHI[3*nv*(3*v+0)+3*index+1] += ptr[i][3*nv*(3*f+0)+3*index+1];
        d_PHI[3*nv*(3*v+0)+3*index+2] += ptr[i][3*nv*(3*f+0)+3*index+2];

        d_PHI[3*nv*(3*v+1)+3*index+0] += ptr[i][3*nv*(3*f+1)+3*index+0];
        d_PHI[3*nv*(3*v+1)+3*index+1] += ptr[i][3*nv*(3*f+1)+3*index+1];
        d_PHI[3*nv*(3*v+1)+3*index+2] += ptr[i][3*nv*(3*f+1)+3*index+2];

        d_PHI[3*nv*(3*v+2)+3*index+0] += ptr[i][3*nv*(3*f+2)+3*index+0];
        d_PHI[3*nv*(3*v+2)+3*index+1] += ptr[i][3*nv*(3*f+2)+3*index+1];
        d_PHI[3*nv*(3*v+2)+3*index+2] += ptr[i][3*nv*(3*f+2)+3*index+2];
      }
    } // endfor
  } // endif 
}

extern "C" {

  void mvc_gpu(scalar_t *d_PHI,
               const scalar_t *d_V,
               const index_t  *d_cageF,
               const scalar_t *d_cageV,
               const index_t nv,
               const index_t ncf,
               const index_t ncv) {
    const unsigned int blocksize = 256;
    const unsigned int numBlocks = (nv+blocksize-1)/blocksize;
    mvc_kernel<<< numBlocks, blocksize >>>
        (d_PHI, d_V, d_cageF, d_cageV, nv, ncf, ncv);
  }

  void green_gpu(scalar_t *d_phix,
                 scalar_t *d_phiy,
                 scalar_t *d_phiz,
                 scalar_t *d_psi,
                 const scalar_t *d_V,
                 const index_t  *d_cageF,
                 const scalar_t *d_cageV,
                 const scalar_t *d_cageN,
                 const index_t nv,
                 const index_t ncf,
                 const index_t ncv,
                 const scalar_t *d_qp,
                 const scalar_t *d_qw,
                 const index_t nq) {
    // parallel through basis columns
    const unsigned int blocksize = 256;
    const unsigned int numBlocks = (ncf*nv+blocksize-1)/blocksize;
    green_kernel<<< numBlocks, blocksize >>>
        (d_phix, d_phiy, d_phiz, d_psi,
         d_V, d_cageF, d_cageV, d_cageN, nv, ncf, ncv,
         d_qp, d_qw, nq);
  }

  void somig_gpu(const scalar_t nu,
                 scalar_t *d_PHIx,
                 scalar_t *d_PHIy,
                 scalar_t *d_PHIz,
                 scalar_t *d_PSI ,
                 const scalar_t *d_V,
                 const index_t  *d_cageF,
                 const scalar_t *d_cageV,
                 const scalar_t *d_cageN,
                 const index_t nv,
                 const index_t ncf,
                 const index_t ncv,
                 const scalar_t *d_qp,
                 const scalar_t *d_qw,
                 const index_t nq) {
    // parallel through basis entries
    const unsigned int blocksize = 256;
    const unsigned int numBlocks = (ncf*nv+blocksize-1)/blocksize;
    somig_kernel<<< numBlocks, blocksize >>>
        (nu, d_PHIx, d_PHIy, d_PHIz, d_PSI,
         d_V, d_cageF, d_cageV, d_cageN, nv, ncf, ncv,
         d_qp, d_qw, nq);
  }

  // reduce phixyz to phi
  void green_gpu_post(scalar_t *d_phi,
                      const scalar_t *d_phix,
                      const scalar_t *d_phiy,
                      const scalar_t *d_phiz,
                      const index_t  *d_cageF,
                      const index_t nv,
                      const index_t ncf,
                      const index_t ncv) {
    // parallel through basis columns
    const unsigned int blocksize = 256;
    const unsigned int numBlocks = (nv+blocksize-1)/blocksize;
    green_kernel_post<<< numBlocks, blocksize >>>
        (d_phi, d_phix, d_phiy, d_phiz, d_cageF, nv, ncf, ncv);
  }

  // reduce PHIxyz to PHI
  void somig_gpu_post(scalar_t *d_PHI,
                      const scalar_t *d_PHIx,
                      const scalar_t *d_PHIy,
                      const scalar_t *d_PHIz,
                      const index_t  *d_cageF,
                      const index_t nv,
                      const index_t ncf) {
    // parallel through basis columns
    const unsigned int blocksize = 256;
    const unsigned int numBlocks = (nv+blocksize-1)/blocksize;
    somig_kernel_post<<< numBlocks, blocksize >>>
        (d_PHI, d_PHIx, d_PHIy, d_PHIz, d_cageF, nv, ncf);
  }
  
}
